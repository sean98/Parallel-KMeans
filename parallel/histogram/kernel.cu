#include "hip/hip_runtime.h"
//Lior Itzhak	203679816
//Sean Goldfarb	209320977
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "Point.h"
#include "Cluster.h"

#define NUM_OF_THREADS 1024
#define NUM_OF_BLOCKS 1

int __device__ atomicAdd(int* address, int val);

__device__ double atomicAddDouble(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

void cudaInit()
{
	hipFree(0);
}

__device__ int getId()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ double cudaDist(vector_t v1, vector_t v2)
{
	return pow(v1.x - v2.x, 2) + pow(v1.y - v2.y, 2) + pow(v1.z - v2.z, 2);
}

__global__ void initPointInCluster(int globalPointInCluster[], int k)
{
	int i = getId();
	if (i<k)
		globalPointInCluster[i] = 0;
}

__global__ void resetClusters(cluster_t clusters[], int k)
{
	int i = getId();
	if (i < k)
	{
		clusters[i].location.x = 0;
		clusters[i].location.y = 0;
		clusters[i].location.z = 0;
	}
}

__global__ void addPointsToCluster(cluster_t clusters[], point_t points[], int pointToCluster[], int n)
{
	int i = getId();
	if (i < n)
	{
		atomicAddDouble(&clusters[pointToCluster[i]].location.x, points[i].location.x);
		atomicAddDouble(&clusters[pointToCluster[i]].location.y, points[i].location.y);
		atomicAddDouble(&clusters[pointToCluster[i]].location.z, points[i].location.z);
	}
}

__global__ void avrageClusters(cluster_t clusters[], int pointsInCluster[], int k)
{
	int i = getId();
	if (i < k)
	{
		clusters[i].location.x /= pointsInCluster[i];
		clusters[i].location.y /= pointsInCluster[i];
		clusters[i].location.z /= pointsInCluster[i];
		pointsInCluster[i] = 0;
	}
}

__global__ void kMeansIteration(cluster_t clusters[], int globalPointInCluster[], int k, point_t points[], int pointToCluster[], int n, int* isSame)
{
	int i = getId();
	if (i < n)
	{
		double minDist = cudaDist(clusters[0].location, points[i].location);
		int minIndex = 0;
		for (int j = 1; j < k; j++)
		{
			double tempDist = cudaDist(clusters[j].location, points[i].location);
			minDist = fmin(minDist, tempDist);
			minIndex = tempDist == minDist ? j : minIndex;
		}
		if (pointToCluster[i] != minIndex) {
			*isSame = 0;
			pointToCluster[i] = minIndex;
		}
		atomicAdd(&globalPointInCluster[minIndex], 1);
	}
}

__global__ void cudaInitClusters(cluster_t clusters[], int k, point_t points[], int n)
{
	int i = getId();
	clusters[i].location.x = points[i].location.x;
	clusters[i].location.y = points[i].location.y;
	clusters[i].location.z = points[i].location.z;
	clusters[i].pointsList = NULL;
}

__global__ void cudaAddPointsParallel(point_t points[], int n, double dt)
{
	int i = getId();
	if (i < n)
	{
		points[i].location.x += dt * points[i].speed.x;
		points[i].location.y += dt * points[i].speed.y;
		points[i].location.z += dt * points[i].speed.z;
	}
}

__global__ void cudaQuality(point_t points[], int pointToCluster[], double maxDist[], int n)
{
	int id = getId();
	if (id < n)
	{
		maxDist[id] = 0;
		for (int i = id + 1; i < n; i++)
		{
			if (pointToCluster[id] == pointToCluster[i])
			{
				double dist = cudaDist(points[id].location, points[i].location);
				maxDist[id] = dist > maxDist[id] ? dist : maxDist[id];
			}
		}
	}
}

void cudaAddPoints(point_t h_dstPoints[], point_t h_srcPoints[], int n, double dt)
{
	point_t* d_points;
	hipMalloc(&d_points, n * sizeof(point_t));
	hipMemcpy(d_points, h_srcPoints, n * sizeof(point_t), hipMemcpyHostToDevice);

	cudaAddPointsParallel<<<n/1024 + 1, 1024>>>(d_points, n, dt);

	hipMemcpy(h_dstPoints, d_points, n * sizeof(point_t), hipMemcpyDeviceToHost);
	hipFree(d_points);
}


cluster_t* CudaKMeans(int n, int k, int limit, point_t h_points[], double* h_qaulity)
{
	//points
	point_t* d_points;
	hipMalloc(&d_points, n * sizeof(point_t));
	hipMemcpy(d_points, h_points, n * sizeof(point_t), hipMemcpyHostToDevice);

	//clusters
	cluster_t* h_clusters = (cluster_t*)calloc(k, sizeof(cluster_t));
	cluster_t* d_clusters;
	hipMalloc(&d_clusters, k * sizeof(cluster_t));
	cudaInitClusters<<<1,k>>>(d_clusters, k, d_points, n);

	//pointToCluster
	int* h_pointToCluster = (int*)calloc(n, sizeof(int));
	int* d_pointToCluster;
	hipMalloc(&d_pointToCluster, n * sizeof(int));

	//pointsInCluster
	int* h_pointsInCluster = (int*)calloc(k, sizeof(int));
	int* d_pointsInCluster;
	hipMalloc(&d_pointsInCluster, k * sizeof(int));

	//maxDist
	double* h_maxDist = (double*)calloc(n, sizeof(double));
	double* d_maxDist;
	hipMalloc(&d_maxDist, n * sizeof(double));

	//isSame
	int one = 1;
	int *d_isSame, h_isSame = 0;
	hipMalloc(&d_isSame, sizeof(int));
	hipMemcpy(d_isSame, &one, sizeof(int), hipMemcpyHostToDevice);

	initPointInCluster <<<1, k >> >(d_pointsInCluster, k);
	for (int i = 0; i < limit && !h_isSame; i++)
	{
		hipMemcpy(d_isSame, &one, sizeof(int), hipMemcpyHostToDevice);
		
		kMeansIteration <<<n/1024 + 1, 1024 >> > (d_clusters, d_pointsInCluster, k, d_points, d_pointToCluster, n, d_isSame);
		resetClusters <<<1, k >> > (d_clusters, k);
		addPointsToCluster<<<n / 1024 + 1, 1024>>>(d_clusters, d_points, d_pointToCluster, n);
		avrageClusters<<<1, k>>>(d_clusters, d_pointsInCluster, k);
		
		hipMemcpy(&h_isSame, d_isSame, sizeof(int), hipMemcpyDeviceToHost);
	}
	hipMemcpy(h_clusters, d_clusters, k * sizeof(cluster_t), hipMemcpyDeviceToHost);
	cudaQuality <<<n / 1024 + 1, 1024 >>>(d_points, d_pointToCluster, d_maxDist, n);
	hipMemcpy(h_pointToCluster, d_pointToCluster, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_maxDist, d_maxDist, n * sizeof(double), hipMemcpyDeviceToHost);
	
	double* radius = (double*)calloc(k, sizeof(double));
	for (int i = 0; i < n; i++)
	{
		//printf_s("%lf\n", sqrt(h_maxDist[i]));
		if (h_maxDist[i] > radius[h_pointToCluster[i]])
			radius[h_pointToCluster[i]] = h_maxDist[i];
	}
	*h_qaulity = 0;
	for (int i = 0; i < k; i++)
	{
		//printf_s("%lf\n", sqrt(radius[i]));
		for (int j = 0; j < k; j++)
		{
			if (i == j)
				continue;
			*h_qaulity += sqrt(radius[i]) / sqrt(distance(h_clusters[i].location, h_clusters[j].location));
		}
	}
	*h_qaulity /= k * (k - 1);
	free(radius);
	free(h_maxDist);
//
//	for (int i = 0; i < k; i++)
//		h_clusters[i].pointsList = NULL;
////		initArrayList(&h_clusters[i].pointsList, n / k);*/
//
//	for (int i = 0; i < n; i++)
//		addElement(&h_clusters[h_pointToCluster[i]].pointsList, &h_points[i]);

	//maxDist
	hipFree(d_maxDist);
	//points
	hipFree(d_points);
	//clusters
	hipFree(d_clusters);
	//pointToCluster
	free(h_pointToCluster);
	hipFree(d_pointToCluster);
	//pointInCluster
	free(h_pointsInCluster);
	hipFree(d_pointsInCluster);
	//isSame
	hipFree(d_isSame);

	return h_clusters;
}